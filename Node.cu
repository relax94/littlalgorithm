#include "hip/hip_runtime.h"
#include "Node.cuh"
#include <iostream>
#include <algorithm>
#include <stack>
#include <vector>
#include "hip/hip_runtime.h"
#include ""

// ------- $TD : 'WRITE MORE ORGANIZED WITH INITIAL FUNCS';
Node::Node(int size, int s, int s0)
{
	this->size = size;


	this->S = s;
	this->S0 = s0;
	this->points = 0;
	

	optimalX = new int[this->size];
	optimalY = new int[this->size];
}


void Node::setInitials(int iSize){
	this->P = new int[iSize];
	for (int i = 0; i < iSize; i++)
		this->P[i] = -1;
}

// --- $ID : 'MAYBE ALLOC MEMORY ON ONE WAY';
void Node::setInitialMatrix(int *sourceMatrix) {
	this->baseSize = this->size;
	this->setInitials(this->baseSize);
	//this->M = new int*[size];
	this->M = (int *)malloc(this->size * this->size * sizeof(int));
	this->translateX = new int[size];
	this->translateY = new int[size];

	for (int i = 0; i < size; i++) {
		//this->M[i] = new int[size];
		this->translateX[i] = i;
		this->translateY[i] = i;
		for (int j = 0; j < size; j++) {
			//this->M[i][j] = sourceMatrix[i][j];
			int offset = i * this->size + j;
			this->M[offset] = sourceMatrix[offset];
		}
	}

}

void Node::setMatrix(int *m) {
	//this->M = new int*[size];
	this->M = (int*)malloc(this->size * this->size * sizeof(int));
	/*for (int i = 0; i < size; i++)
		this->M[i] = new int[size];*/

	for (int i = 0; i < size; i++) {
		//this->M[i] = new int[size];
		for (int j = 0; j < size; j++) {

			int offset = i * this->size + j;
			this->M[offset] = m[offset];
		}
	}
}

// ---- $TD: 'REWRITE WITH MORE PRODUCITY BY BINARY COPYING'
void Node::setMatrixWithRemoveExclude(int *source, int row, int col) {

	this->M = (int *)malloc(this->size * this->size * sizeof(int));
	int originalSize = this->size + 1;

	int ni = 0;

	for (int i = 0; i < originalSize; i++){
		if (i != row){
			for (int j = 0; j < originalSize; j++){
				if (j != col){
					this->M[ni] = source[i * originalSize + j];
					ni++;
				}
			}
		}
	}

}

Node::~Node()
{

}

void Node::printMatrix() {
	std::cout << std::endl;
	std::cout << std::endl;

	for (int i = 0; i < this->size; i++) {
		for (int j = 0; j < this->size; j++)
			std::cout << this->M[i * this->size + j] << " ";
		std::cout << std::endl;
	}

	std::cout << std::endl;
	std::cout << std::endl;
}

// $TD: 'REPLACE BY STANDART STD OR BOOST';
int Node::getArrayMinValue(int restrictVal, /*int *row*/ int row) {
	int min = InfityMaxValue;
	for (int i = 0; i < size; i++) {
		int offset = row * this->size + i;
		if (this->M[offset] > restrictVal && this->M[offset] < min)
			min = this->M[offset];
	}
	return min;
}

// $TD : 'REWRITE BY INIT WAY : CHANGES 4 CYCLES BY 2 AND
//								SPEED UP  getPathForRemove ---> BY INDEXING OPERATION
//								SPEED UP  subMRows ---> return this.minRowsEls[row];
//								SPPED UP  subMCols ---> return this.minRowsEls[col] - this.minRowsEls[row] - checkin;
void Node::subMinRowsAndCorrect() {

	//this->printMatrix();

	//int matrixSize = this->size * this->size;
	//int allocatedSize = matrixSize * sizeof(int);

	//int *d_m;
	//int *d_t;

	//int s = 100;
	//int *d_s;
	//int *d_a;

	//hipMalloc((void**)&d_s, this->size * sizeof(int));
	//hipMalloc((void**)&d_a, this->size * this->size * sizeof(int));
	/*hipMalloc((void**)&d_m, allocatedSize);
	hipMalloc((void**)&d_t, this->size * sizeof(int));

	hipMemcpy(d_m, this->M, allocatedSize, hipMemcpyHostToDevice);
	hipMemcpy(d_s, &s, sizeof(int), hipMemcpyHostToDevice);

	matrixRowCorrect << <this->size, this->size >> >(d_m, d_t, d_s);

	int *response = (int*)malloc(allocatedSize);
	int *temp = (int*)malloc(this->size * sizeof(int));
	hipMemcpy(response, d_m, allocatedSize, hipMemcpyDeviceToHost);
	hipMemcpy(temp, d_t, this->size * sizeof(int), hipMemcpyDeviceToHost);
	*/
	//this->testMatrixAdduction(this->M);

	/* PREV STABLE VERSION*/

	//int *mins = new int[this->size];
	//int localSDelta = 0;
	//int min = InfityMaxValue;
	//for (int i = 0; i < size; i++) {
	//	min = getArrayMinValue(-1, i);
	//	localSDelta += min < InfityMaxValue ? min : 0;
	//	mins[i] = min;
	//	for (int j = 0; j < size; j++) {
	//		int offset = i * this->size + j;
	//		if (this->M[offset] > -1)
	//			this->M[offset] -= min;
	//		/*if (this->M[i][j] > -1)
	//			this->M[i][j] -= min;*/
	//	}
	//	//r = subMinRowsAndCorrect(min, this->M[i], this->size);
	//}
	//S += localSDelta;


	int *mins = new int[this->size];
	int localSDelta = 0;
	int min = InfityMaxValue;
	for (int i = 0; i < size; i++) {
		min = getArrayMinValue(-1, i);
		localSDelta += min < InfityMaxValue ? min : 0;
		mins[i] = min;
		if (min > 0){
			for (int j = 0; j < size; j++) {
				int offset = i * this->size + j;
				if (this->M[offset] > -1)
					this->M[offset] -= min;
				/*if (this->M[i][j] > -1)
				this->M[i][j] -= min;*/
			}
		}
		//r = subMinRowsAndCorrect(min, this->M[i], this->size);
	}
	S += localSDelta;

	/*hipMemcpy(d_s, mins, this->size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_a, this->M, this->size * this->size * sizeof(int), hipMemcpyHostToDevice);
	arrayReduce << <this->size, this->size >> >(d_a, d_s);
	hipMemcpy(this->M, d_a, this->size * this->size *  sizeof(int), hipMemcpyDeviceToHost);*/

	//	this->printMatrix();

	/*hipFree(d_a);
	hipFree(d_s);*/

	//this->printMatrix();
}

// REWRITE BY subMinRowsAndCorrect
void Node::subMinColsAndCorrect() {
	int correlation = 0;
	int localMin = InfityMaxValue;
	/*int *colMins = new int[size];*/
	int offsetj = 0;

	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			 offsetj = i + this->size * j;
			if (this->M[offsetj] < localMin && this->M[offsetj] > -1)
				localMin = this->M[offsetj];
		}
		//colMins[i] = localMin;

		if (localMin != 0){
			for (int j = 0; j < size; j++)
				if (this->M[offsetj] > -1)
					this->M[offsetj] -= localMin;
		}


		correlation += localMin == InfityMaxValue ? 0 : localMin;
		localMin = InfityMaxValue;
	}

	/*for (int i = 0; i < size; i++)
	if (colMins[i] != 0){
		for (int j = 0; j < size; j++)
			if (this->M[i + this->size * j] > -1)
				this->M[i + this->size * j] -= colMins[i];
			}*/


	S += correlation;
}

// REWRITE WITH PREVIOS COMMENT LOGIC : SPEED UP
void Node::getPathForRemove(int &rowE, int &colE) {
	int max = -1;
	int rowMin = InfityMaxValue;
	int colMin = InfityMaxValue;
	int ioffset, loffset;

	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
				ioffset = i*this->size;
			if (this->M[ioffset + j] == 0) {

				rowMin = InfityMaxValue;
				colMin = InfityMaxValue;

				for (int r = 0; r < size; r++) {
					if (this->M[ioffset + r] != -1 && j != r && this->M[ioffset + r] < rowMin) {
						rowMin = this->M[ioffset + r];
					}

						loffset = r * this->size;
					if (this->M[loffset + j] != -1 && r != i && this->M[loffset + j] < colMin) {
						colMin = this->M[loffset + j];
					}

				}
			
				if ((colMin + rowMin) > max)
				{
					max = colMin + rowMin;
					rowE = i;
					colE = j;
				}
			}
		}
	}
}

void Node::invokeAdduction() {
	this->subMinRowsAndCorrect();
	this->subMinColsAndCorrect();
}

void Node::copySessionDescription(int *rowD, int *colD) {
	this->translateX = new int[this->size];
	this->translateY = new int[this->size];



	int ri = 0;
	int rj = 0;
	for (int i = 0; i < this->size + 1; i++)
	{
		if (rowD[i] != -1){
			this->translateX[ri] = rowD[i];
			ri++;
		}

		if (colD[i] != -1){
			this->translateY[rj] = colD[i];
			rj++;
		}
	}
}


Node* Node::leftBranching(int row, int col) {
	Node *node = new Node(this->size, this->S, this->S);
	node->baseSize = this->baseSize;

	node->cudaCleanCopy(this->M);
	//node->setMatrix(this->M);

	node->copySessionDescription(this->translateX, this->translateY);
	node->M[row * this->size + col] = -1;
	node->invokeAdduction();

	//node->printMatrix();

	node->setInitials(this->baseSize);
	//for (int i = 0; i < this->baseSize; i++) {
	//	if (this->P[i] != -1)
	//		node->P[i] = this->P[i];
	//	else
	//		node->P[i] = -1;
	//}

	memcpy(node->P, this->P, this->baseSize * sizeof(int));

	return node;
}

int Node::getRealElement(int *dataDescription, int ind) {

	for (int i = 0; i < this->size; i++)
	{
		if (dataDescription[i] == ind)
			return i;
	}
	return -1;
}

void Node::printArray(int size, int *arr){
	std::cout << std::endl;
	std::cout << std::endl;
	for (int i = 0; i < size; i++)
		std::cout << arr[i] << std::endl;
	std::cout << std::endl;
	std::cout << std::endl;
}

// REMOVE SOME FUNCTIONS
// $DIRTY CODE
Node* Node::rightBranching(int row, int col) {
	Node *node = new Node(this->size - 1, this->S, this->S);
	node->baseSize = this->baseSize;
	node->setInitials(this->baseSize);

	int realRow = this->translateX[row];
	int realCol = this->translateY[col];

	this->P[realRow] = realCol;

	memcpy(node->P, this->P, this->baseSize * sizeof(int));

	//printArray(this->baseSize, node->P);

	
	this->points++;
	node->points = this->points;


	///	printArray(this->baseSize, node->P);


	if (realRow > -1 && realCol > -1) {
		if (points > 1)
			node->handlePodcycles(realRow, realCol);

		/*printArray(this->size, translateX);*/

		//auto tmp = *std::max_element(this->translateX, this->translateX + sizeof(this->translateX) / sizeof(int));

		if (row < this->size && col < this->size){
			this->translateX[row] = -1;
			this->translateY[col] = -1;
		}

		/*this->printArray(this->size, this->translateX);
		this->printArray(this->size, this->translateY);*/

		int returnRow = getRealElement(this->translateX, realCol);
		int returnCol = getRealElement(this->translateY, realRow);

		if (returnRow > -1 && returnCol > -1)
			this->M[returnRow * this->size + returnCol] = -1;

		node->setMatrixWithRemoveExclude(this->M, row, col);

		//node->printMatrix();

		node->invokeAdduction();

		//node->printMatrix();

		node->copySessionDescription(this->translateX, this->translateY);

	}

	return node;
}

void Node::handleStraightforwardMatrix() {
	for (int i = 0; i < this->size; i++) {
		if (P[i] == -1) {
			int offsetX = i*this->size;
			for (int j = 0; j < this->size; j++)
			if (this->M[offsetX + j] == 0)
				this->P[i] = j;
		}
	}
}

/* --> NO USE IN PRODUCTION !!! <-------------------- HANDLE LOCAL CYCLES (REWRITE) $DIRTY CODE ----------------*/
int Node::getHead(int tail) {
	for (int i = 0; i < this->baseSize; i++) {
		if (this->P[i] == tail)
			return i;
	}
	return -1;
}

int Node::getTail(int head) {
	return this->P[head];
}

void Node::handlePodcycles(int &a, int &b) {
	int source = a;
	int destiny = b;


	std::vector<int> localCycle;
	int head = 0;
	int tail = 0;
	bool finish = false;

	localCycle.push_back(a);
	localCycle.push_back(b);
	int countIterations = 0;
	while (!finish)
	{
		countIterations++;
		//if (localCycle.size() > this->baseSize + 10)
		//{
		//	std::cout << "CYCLE ERROR !!!!!!" << std::endl;
		//	throw "ss";
		//}

		if (head != -1) {
			head = getHead(a);
			if (b == head) // whaaaat ?
				break;
			if (head != -1) {
				localCycle.insert(localCycle.begin(), head);
				a = head;
			}
		}

		if (tail != -1) {
			tail = getTail(b);
			if (tail == head) // whaaaat ?
				break;
			if (tail != -1) {
				localCycle.push_back(tail);
				b = tail;
			}
		}

		if (head == -1 && tail == -1)
			finish = true;
	}
	a = localCycle.front();
	b = localCycle.back();
	localCycle.clear();
}





__global__ void modifyArrayKernel(int *val, int *arr){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < 6 && arr[i] > -1)
		arr[i] = arr[i] - *val;
}

#define N 2

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
	int i = threadIdx.x;
	int j = threadIdx.y;

	C[i][j] = A[i][j] + B[i][j];
}

__global__ void testMatrix(int **M, int **R){
	int i = threadIdx.x;
	int j = threadIdx.y;

	R[i][j] = M[i][j] - 10;
}

__global__ void testKernel(int *s, const int *re){

	__shared__ int temp[1];

	int i = threadIdx.x;
	if (re[i] > -1 && re[i] < temp[0])
		temp[0] = re[i];

	__syncthreads();

	*s = temp[0];
}

void Node::testMatrixAdduction(int *M){


	int A[N][N] = { { 1, 2 }, { 3, 4 } };
	int B[N][N] = { { 5, 6 }, { 7, 8 } };
	int C[N][N] = { { 0, 0 }, { 0, 0 } };

	int(*pA)[N], (*pB)[N], (*pC)[N];

	hipMalloc((void**)&pA, (N*N)*sizeof(int));
	hipMalloc((void**)&pB, (N*N)*sizeof(int));
	hipMalloc((void**)&pC, (N*N)*sizeof(int));

	hipMemcpy(pA, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pB, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pC, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = 1;
	dim3 threadsPerBlock(N, N);
	MatAdd << <numBlocks, threadsPerBlock >> >(pA, pB, pC);

	hipMemcpy(C, pC, (N*N)*sizeof(int), hipMemcpyDeviceToHost);




	/*int **R = new int*[this->size];
	for (int i = 0; i < this->size; i++)
	R[i] = new int[this->size];

	int **dev_M = (int **)malloc(this->size * sizeof(int*));
	int **dev_R = (int **)malloc(this->size * sizeof(int*));

	for (int i = 0; i < this->size; i++){
	dev_M[i] = (int *)malloc(this->size * sizeof(int));
	dev_R[i] = (int *)malloc(this->size * sizeof(int));
	}

	int size = (this->size * this->size) * sizeof(int);

	hipMalloc((void**)dev_M, size);
	hipMalloc((void**)dev_R, size);

	testMatrix << <1, this->size >> >(dev_M, dev_R);


	hipMemcpy(R, dev_R, size, hipMemcpyDeviceToHost);*/

}

int* Node::subMinRowsAndCorrect(int s, const int *row, const int size){


	int *dev_s = 0;
	int *dev_re = 0;
	int *arr = (int *)malloc(size * sizeof(int));
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus == hipSuccess){
		hipMalloc((void**)&dev_s, sizeof(int));
		hipMalloc((void**)&dev_re, size * sizeof(int));

		hipMemcpy(dev_s, &s, sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_re, row, size * sizeof(int), hipMemcpyHostToDevice);

		//testKernel <<<1, size >>>(dev_s, dev_re);

		modifyArrayKernel << <1, size >> >(dev_s, dev_re);

		hipDeviceSynchronize();

		int *c = (int *)malloc(sizeof(int));


		hipMemcpy(c, dev_s, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(arr, dev_re, size * sizeof(int), hipMemcpyDeviceToHost);

		hipFree(dev_s);
		hipFree(dev_re);
	}

	return arr;
}

__global__ void minValue(int *source, int *val){
	__shared__ int temp[1];

	int currentValue = source[threadIdx.x];
	if (currentValue > -1 && currentValue < *val){
		temp[0] = currentValue;
	}

	__syncthreads();

	*val = temp[0];
}

__device__ int minVal = 100;

__device__ int blockChange = 0;

__global__ void matrixRowCorrect(int *arr, int *tmp, int *s){

	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (blockIdx.x != blockChange){
		blockChange = blockIdx.x;
		minVal = 100;
	}

	int currentValue = arr[id];

	if (currentValue < minVal){
		tmp[blockIdx.x] = currentValue;
		minVal = currentValue;
	}

}

__global__ void arrayReduce(int *m, int *ms){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (m[id] > -1)
		m[id] = m[id] - ms[blockIdx.x];
}

__global__ void cleanCopy(int *S, int *D){
	D[threadIdx.x] = S[threadIdx.x];
}

int * Node::cudaCleanCopy(int *source){

	int matrixSize = this->size * this->size;
	int allocatedSize = matrixSize * sizeof(int);

	//int *d_source = (int*)malloc(allocatedSize);
	//int *d_destiny = (int*)malloc(allocatedSize);

	////memcpy(d_source, source, allocatedSize);

	//hipMalloc((void**)&d_source, allocatedSize);
	//hipMalloc((void**)&d_destiny, allocatedSize);

	//hipMemcpy(d_source, source, allocatedSize, hipMemcpyHostToDevice);

	//cleanCopy << <1, matrixSize >> >(d_source, d_destiny);

	this->M = (int*)malloc(allocatedSize);

	//hipMemcpy(this->M, d_destiny, allocatedSize, hipMemcpyDeviceToHost);

	//hipFree(d_destiny);
	//hipFree(d_source);
	//free(source);

	memcpy(this->M, source, allocatedSize);

	return this->M;
}

